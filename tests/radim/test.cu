
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1

__global__ void mykernel(int *a, int *b, int *c)
{
   *c = *a + *b;
}

int main()
{
   int *host_a = (int*)malloc(N * sizeof(int));
   int *host_b = (int*)malloc(N * sizeof(int));
   int *host_c = (int*)malloc(N * sizeof(int));
   int *gpu_a = NULL;
   int *gpu_b = NULL;
   int *gpu_c = NULL;
   hipMalloc(&gpu_a, N * sizeof(int));
   hipMalloc(&gpu_b, N * sizeof(int));
   hipMalloc(&gpu_c, N * sizeof(int));

   hipMemcpy(gpu_a, host_a, N * sizeof(int), hipMemcpyHostToDevice);
   hipMemcpy(gpu_b, host_b, N * sizeof(int), hipMemcpyHostToDevice);

	mykernel<<<1,1>>>(gpu_a, gpu_b, gpu_c);

   hipMemcpy(host_c, gpu_c, N * sizeof(int), hipMemcpyDeviceToHost);

   hipFree(gpu_a);
   hipFree(gpu_b);
   hipFree(gpu_c);
   free(host_a);
   free(host_b);
   free(host_c);

	return 0;
}
